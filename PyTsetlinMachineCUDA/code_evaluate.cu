#include <hip/hip_runtime.h>



#define CLASSES 1
#define CLAUSES 1
#define FEATURES 1
#define STATE_BITS 1
#define BOOST_TRUE_POSITIVE_FEEDBACK 1
#define S 1
#define THRESHOLD 1
#define Q 1

#define NEGATIVE_CLAUSES 1

#define PATCHES 1

#define NUMBER_OF_EXAMPLES 1

#include <hiprand/hiprand_kernel.h>
#define INT_SIZE 32

#define LA_CHUNKS (((FEATURES-1)/INT_SIZE + 1))
#define CLAUSE_CHUNKS ((CLAUSES-1)/INT_SIZE + 1)

#if (FEATURES % 32 != 0)
#define FILTER (~(0xffffffff << (FEATURES % INT_SIZE)))
#else
#define FILTER 0xffffffff
#endif


extern "C"
{
    // Evaluate examples
    __global__ void evaluate(unsigned int *global_ta_state, int *clause_weights, int *class_sum, int *X)
    {
        int index = blockIdx.x * blockDim.x + threadIdx.x;
        int stride = blockDim.x * gridDim.x;

        for (int clause = index; clause < CLAUSES; clause += stride) {
            
            //unsigned int type_clause;
			//#if -1 == 1 - 2 * (clause % 2)
			//	type_clause = 0;
			//#else
			//	type_clause = 1;
			//#endif
            
            
            
            unsigned int *ta_state = &global_ta_state[clause*LA_CHUNKS*STATE_BITS];

            int all_exclude = 1;
            for (int la_chunk = 0; la_chunk < LA_CHUNKS-1; ++la_chunk) {
                if (ta_state[la_chunk*STATE_BITS + STATE_BITS - 1] > 0) {
                    all_exclude = 0;
                    break;
                }
            }

            if ((ta_state[(LA_CHUNKS-1)*STATE_BITS + STATE_BITS - 1] & FILTER) > 0) {
                all_exclude = 0;
            }

            if (all_exclude) {
                continue;
            }

            for (int e = 0; e < NUMBER_OF_EXAMPLES; ++e) {
                int clause_output;
                
                int sum_over_patches = 0; 
                for (int patch = 0; patch < PATCHES; ++patch) {
                    clause_output = 1;
                    for (int la_chunk = 0; la_chunk < LA_CHUNKS-1; ++la_chunk) {
                        if ((ta_state[la_chunk*STATE_BITS + STATE_BITS - 1] & X[e*(LA_CHUNKS*PATCHES) + patch*LA_CHUNKS + la_chunk]) != ta_state[la_chunk*STATE_BITS + STATE_BITS - 1]) {
                            clause_output = 0;
                            break;
                        }
                    }

                    if ((ta_state[(LA_CHUNKS-1)*STATE_BITS + STATE_BITS - 1] & X[e*(LA_CHUNKS*PATCHES) + patch*LA_CHUNKS + LA_CHUNKS-1] & FILTER) != (ta_state[(LA_CHUNKS-1)*STATE_BITS + STATE_BITS - 1] & FILTER)) {
                        clause_output = 0;
                    }
                    sum_over_patches += clause_output;
                }

                //printf(" %d \\n", sum_over_patches);
                if (sum_over_patches == 2) {
                    int class_id = clause / (CLAUSES / CLASSES);

                    int clause_weight = clause_weights[clause];
                    atomicAdd(&class_sum[class_id*NUMBER_OF_EXAMPLES + e], clause_weight);
                }
                else if (sum_over_patches == 1) {
                    int class_id = clause / (CLAUSES / CLASSES);

                    int clause_weight = clause_weights[clause];
                    //printf(" %d \\n", clause_weight);
                    atomicAdd(&class_sum[class_id*NUMBER_OF_EXAMPLES + e], -clause_weight);
                }


            }
        }
    }
}